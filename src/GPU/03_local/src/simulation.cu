#include "hip/hip_runtime.h"
#include "simulation.cuh"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "common.cuh"

__global__ void DiffuseKnl(
    double *__restrict__ tnow,
    double *__restrict__ tnext,
    double cx,
    double cy,
    double cz) {
  extern __shared__ double sdata[];
  const int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  const int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
  if (i - 1 < IMAX - 2 && j - 1 < JMAX - 2 && k - 1 < KMAX - 2) {
    const int jstride = blockDim.x + 2;
    const int kstride = (blockDim.x + 2) * (blockDim.y + 2);
    const int il = threadIdx.x + 1;
    const int jl = threadIdx.y + 1;
    const int kl = threadIdx.z + 1;
    const int center = kl * kstride + jl * jstride + il;
    sdata[center] = tnow[INDEX3D(i, j, k)];
    if (threadIdx.x == 0) {
      sdata[center-1] = tnow[INDEX3D(i-1, j, k)];
    }
    if (threadIdx.x == blockDim.x-1 || i == IMAX-2) {
      sdata[center+1] = tnow[INDEX3D(i+1, j, k)];
    }
    if (threadIdx.y == 0) {
      sdata[center-jstride] = tnow[INDEX3D(i, j-1, k)];
    } 
    if (threadIdx.y == blockDim.y-1 || j == JMAX-2) {
      sdata[center+jstride] = tnow[INDEX3D(i, j+1, k)];
    }
    if (threadIdx.z == 0) {
      sdata[center-kstride] = tnow[INDEX3D(i, j, k-1)];
    } 
    if (threadIdx.z == blockDim.z-1 || k == KMAX-2) {
      sdata[center+kstride] = tnow[INDEX3D(i, j, k+1)];
    }
    __syncthreads();
     // Diffuse
    tnext[INDEX3D(i, j, k)] = sdata[center] + cx * (sdata[center-1] - 2.0*sdata[center] + sdata[center+1])
                                            + cy * (sdata[center-jstride] - 2.0*sdata[center] + sdata[center+jstride])
                                            + cz * (sdata[center-kstride]- 2.0*sdata[center] + sdata[center+kstride]);
  }
}

// Note, i and j here are not tied to the 3d coordinate system
// They are offsets into a 2d plane, which can be oriented normal 
// to any of the 3d i,j,k axis (so 2d i, j can represent any 3d axis)
__global__ void ReflectKnl(double *__restrict__ tnext,
                           int offset, // First (lowest) inner cell gid
                           int istride, int ispan,
                           int jstride, int jspan, 
                           int shift) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < ispan * jspan) {
    const int i = tid % ispan;
    const int j = tid / ispan;
    const int inner = offset + i * istride + j * jstride;
    const int outer = inner + shift;
    tnext[outer] = tnext[inner];
  }
}
