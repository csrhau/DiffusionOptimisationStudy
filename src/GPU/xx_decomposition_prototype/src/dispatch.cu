#include "hip/hip_runtime.h"
#include "dispatch.cuh"

#include "common.h"

#include "kernels.cuh"

void LaunchSingleIteration(double *__restrict__ state_now,
                           double *__restrict__ state_next,
                           int ts,
                           int i0, int i1, int imax,
                           int j0, int j1, int jmax,
                           int k0, int k1, int kmax,
                           double cx,
                           double cy,
                           double cz) {
  double *device_now, *device_next;
  hipMalloc((void **) &device_now, imax * jmax * kmax * sizeof(double));
  hipMalloc((void **) &device_next, imax * jmax * kmax * sizeof(double));
  hipMemcpy(device_now, state_now, imax * jmax * kmax * sizeof(double), hipMemcpyHostToDevice);
  int tpb = (imax * jmax * kmax + 127) / 128;
  DiffuseReflectKnl<<<tpb, 128>>>(device_now, device_next,
                                  ts,
                                  i0, i1, imax,
                                  j0, j1, jmax,
                                  k0, k1, kmax,
                                  cx, cy, cz);
  // Run single iteration
  hipMemcpy(state_next, device_next, imax * jmax * kmax * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(device_now);
  hipFree(device_next);
}
